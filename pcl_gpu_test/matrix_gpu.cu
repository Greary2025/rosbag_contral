#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
  // 添加这个头文件

using namespace std;

// CUDA kernel for matrix multiplication
__global__ void matrixMul(const double* a, const double* b, double* c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n && col < n) {
        double sum = 0.0;
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

// 生成随机数
double GenerateRandomRealValue() {
    std::random_device rd;
    std::default_random_engine eng(rd());
    std::uniform_real_distribution<double> distr(1, 10);
    return distr(eng);
}

int main() {
    const int n = 3;  // 矩阵大小
    const int size = n * n * sizeof(double);

    // 主机内存分配
    double *a = new double[n * n];
    double *b = new double[n * n];
    double *c = new double[n * n];

    // 初始化矩阵
    for (int i = 0; i < n * n; i++) {
        a[i] = GenerateRandomRealValue();
        b[i] = GenerateRandomRealValue();
    }

    // GPU设备内存
    double *da, *db, *dc;
    
    auto t_begin = std::chrono::high_resolution_clock::now();
    std::chrono::high_resolution_clock::time_point t1, t2;
    std::chrono::duration<double, std::milli> time_span;

    t1 = std::chrono::high_resolution_clock::now();
    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dc, size);
    t2 = std::chrono::high_resolution_clock::now();
    time_span = t2 - t1;
    std::cout << "GPU malloc takes " << time_span.count() << " ms\n";

    t1 = std::chrono::high_resolution_clock::now();
    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);
    t2 = std::chrono::high_resolution_clock::now();
    time_span = t2 - t1;
    std::cout << "hipMemcpy takes " << time_span.count() << " ms\n";

    t1 = std::chrono::high_resolution_clock::now();
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (n + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matrixMul<<<numBlocks, threadsPerBlock>>>(da, db, dc, n);
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    time_span = t2 - t1;
    std::cout << "gpu takes " << time_span.count() << " ms\n";

    t1 = std::chrono::high_resolution_clock::now();
    hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);
    t2 = std::chrono::high_resolution_clock::now();
    time_span = t2 - t1;
    std::cout << "hipMemcpy back takes " << time_span.count() << " ms\n";

    // 清理内存
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    delete[] a;
    delete[] b;
    delete[] c;

    auto t_end = std::chrono::high_resolution_clock::now();
    time_span = t_end - t_begin;
    std::cout << "GPU total takes " << time_span.count() << " ms\n";

    return 0;
}
